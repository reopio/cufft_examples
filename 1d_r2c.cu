#include <stdio.h>
#include <stdlib.h>
//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>


#define LENGTH 8
int main()
{

  int nDevices;
  //hipDeviceProp_t
  hipGetDeviceCount(&nDevices);
  
  printf("Number of devices: %d\n", nDevices);
  
  for (int i = 0; i < nDevices; i++) {
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (MHz): %d\n",
           prop.memoryClockRate/1024);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
    printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
    printf("  minor-major: %d-%d\n", prop.minor, prop.major);
    printf("  Warp-size: %d\n", prop.warpSize);
    printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
    printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
  }

    float Data[LENGTH] = {2,1,-1,5,0,3,0,-4};
    hipfftReal *CompData=(hipfftReal*)malloc(LENGTH*sizeof(hipfftReal));
    hipfftComplex *CompData_C=(hipfftComplex*)malloc(LENGTH*sizeof(hipfftComplex));
    int i;
    for(i=0;i<LENGTH;i++)
    {
        CompData[i]=Data[i];
    }

    hipfftReal *i_fftData,*i_fftData_out;
    hipfftComplex *o_fftData;
    hipMalloc((void**)&i_fftData,LENGTH*sizeof(hipfftReal));
    hipMalloc((void**)&i_fftData_out,LENGTH*sizeof(hipfftReal));
    hipMalloc((void**)&o_fftData,LENGTH*sizeof(hipfftComplex));
    hipMemcpy(i_fftData,CompData,LENGTH*sizeof(hipfftReal),hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan,LENGTH,HIPFFT_R2C,1);
    hipfftExecR2C(plan,(hipfftReal*)i_fftData,(hipfftComplex*)o_fftData);
    hipDeviceSynchronize();
    hipMemcpy(CompData_C,o_fftData,LENGTH*sizeof(hipfftComplex),hipMemcpyDeviceToHost);


  printf("\nfft:\n");
  printf("--------------------\n");
  for(i=0;i<LENGTH;i++)
  {

    printf("%.3f",CompData_C[i].x);

    if(CompData_C[i].y != 0.0 )
    {
      printf("+%.3fi",CompData_C[i].y);
    } 
    printf("\n");
  }

  hipMemset(CompData,0,LENGTH*sizeof(hipfftReal));

  hipfftPlan1d(&plan,LENGTH,HIPFFT_C2R,1);
  hipfftExecC2R(plan,(hipfftComplex*)o_fftData,(hipfftReal*)i_fftData_out);
  hipDeviceSynchronize();
  hipMemcpy(CompData,i_fftData_out,LENGTH*sizeof(hipfftReal),hipMemcpyDeviceToHost);

  printf("\n\ninverse fft:\n");
  printf("--------------------\n");
  for(i=0;i<LENGTH;i++)
  {
    printf("%.3f",CompData[i]/8.0);

    if(CompData[i] != 0 )
    {
      printf("+%.3fi",CompData[i]/8.0);
    } 
    printf("\n");
  }

     hipfftDestroy(plan);
     free(CompData);
     free(CompData_C);
     hipFree(o_fftData);
     hipFree(i_fftData);
     hipFree(i_fftData_out);

}