#include <stdio.h>
#include <stdlib.h>
//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>


#define LENGTH 8
int main()
{

  int nDevices;
  //hipDeviceProp_t
  hipGetDeviceCount(&nDevices);
  
  printf("Number of devices: %d\n", nDevices);
  
  for (int i = 0; i < nDevices; i++) {
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (MHz): %d\n",
           prop.memoryClockRate/1024);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
    printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
    printf("  minor-major: %d-%d\n", prop.minor, prop.major);
    printf("  Warp-size: %d\n", prop.warpSize);
    printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
    printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
  }

    float Data[LENGTH] = {2,1,-1,5,0,3,0,-4};
    hipfftComplex *CompData=(hipfftComplex*)malloc(LENGTH*sizeof(hipfftComplex));
    int i;
    for(i=0;i<LENGTH;i++)
    {
        CompData[i].x=Data[i];
        CompData[i].y=0;
    }

    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,LENGTH*sizeof(hipfftComplex));
    hipMemcpy(d_fftData,CompData,LENGTH*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan,LENGTH,HIPFFT_C2C,1);
    hipfftExecC2C(plan,(hipfftComplex*)d_fftData,(hipfftComplex*)d_fftData,HIPFFT_FORWARD);
    hipDeviceSynchronize();
    hipMemcpy(CompData,d_fftData,LENGTH*sizeof(hipfftComplex),hipMemcpyDeviceToHost);


  printf("\nfft:\n");
  printf("--------------------\n");
  for(i=0;i<LENGTH;i++)
  {

    printf("%.3f",CompData[i].x);

    if(CompData[i].y != 0.0 )
    {
      printf("+%.3fi",CompData[i].y);
    } 
    printf("\n");
  }

  hipfftExecC2C(plan,(hipfftComplex*)d_fftData,(hipfftComplex*)d_fftData,HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  hipMemcpy(CompData,d_fftData,LENGTH*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

  printf("\n\ninverse fft:\n");
  printf("--------------------\n");
  for(i=0;i<LENGTH;i++)
  {
    printf("%.3f",CompData[i].x/8.0);

    if(CompData[i].y != 0 )
    {
      printf("+%.3fi",CompData[i].y/8.0);
    } 
    printf("\n");
  }

     hipfftDestroy(plan);
     free(CompData);
     hipFree(d_fftData);

}