#include <stdio.h>
#include <stdlib.h>
//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>

#define N 4
#define LENGTH 16
int main()
{

  int nDevices;
  //hipDeviceProp_t
  hipGetDeviceCount(&nDevices);
  
  printf("Number of devices: %d\n", nDevices);
  
  for (int i = 0; i < nDevices; i++) {
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (MHz): %d\n",
           prop.memoryClockRate/1024);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
    printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
    printf("  minor-major: %d-%d\n", prop.minor, prop.major);
    printf("  Warp-size: %d\n", prop.warpSize);
    printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
    printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
  }

    float Data[LENGTH] = {1,2,3,6,
    3,4,5,9,
    4,5,6,4,
    4,3,6,9};
    hipfftComplex *CompData=(hipfftComplex*)malloc(LENGTH*sizeof(hipfftComplex));
    int i;
    for(i=0;i<LENGTH;i++)
    {
        CompData[i].x=Data[i];
        CompData[i].y=0;
    }

    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,LENGTH*sizeof(hipfftComplex));
    hipMemcpy(d_fftData,CompData,LENGTH*sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan2d(&plan,N,N,HIPFFT_C2C);
    hipfftExecC2C(plan,(hipfftComplex*)d_fftData,(hipfftComplex*)d_fftData,HIPFFT_FORWARD);
    hipDeviceSynchronize();
    hipMemcpy(CompData,d_fftData,LENGTH*sizeof(hipfftComplex),hipMemcpyDeviceToHost);


  printf("\nfft:\n");
  printf("--------------------\n");
  for(i=0;i<LENGTH;i++)
  {

    printf("%.3f",CompData[i].x);

    if(CompData[i].y != 0.0 )
    {
      printf("+%.3fi",CompData[i].y);
    } 
    if((i+1)%N==0){
      printf("\n");
    }else{
      printf(", ");
    }
  }

  hipfftExecC2C(plan,(hipfftComplex*)d_fftData,(hipfftComplex*)d_fftData,HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  hipMemcpy(CompData,d_fftData,LENGTH*sizeof(hipfftComplex),hipMemcpyDeviceToHost);

  printf("\n\ninverse fft:\n");
  printf("--------------------\n");
  for(i=0;i<LENGTH;i++)
  {
    printf("%.3f",CompData[i].x/LENGTH);

    if(CompData[i].y != 0.0 )
    {
      printf("+%.3fi",CompData[i].y/LENGTH);
    } 
    if((i+1)%N==0){
      printf("\n");
    }else{
      printf(", ");
    }
  }
  printf("\n");

     hipfftDestroy(plan);
     free(CompData);
     hipFree(d_fftData);

}